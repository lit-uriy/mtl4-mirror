// Software License for MTL
//
// Copyright (c) 2007 The Trustees of Indiana University.
//               2008 Dresden University of Technology and the Trustees of Indiana University.
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
//
// This file is part of the Matrix Template Library
//
// See also license.mtl.txt in the distribution.


#ifndef MTL_CUDA_DEVICE_VECTOR_NEW_INCLUDE
#define MTL_CUDA_DEVICE_VECTOR_NEW_INCLUDE

#include <stdio.h>

namespace mtl { namespace cuda {


template <typename T>
T* device_vector_new(const int n)
{
    T* pointer;
    hipMalloc(reinterpret_cast<void **>(&pointer), sizeof(T)*n);
    return pointer;
}

template <typename T>
T* device_vector_new(const T& value, const int n)
{
    T* pointer= device_vector_new<T>(n);

    // copy value to first entry and replicate it
    hipMemcpy(pointer, &value, sizeof(T), hipMemcpyHostToDevice);
    for (int i = 1; i < n; i++)
	hipMemcpy(pointer + i, pointer, sizeof(T), hipMemcpyDeviceToDevice);

    return pointer;
}


}} // namespace mtl::cuda

#endif // MTL_CUDA_DEVICE_NEW_INCLUDE

