// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University. 
//               2008 Dresden University of Technology and the Trustees of Indiana University. 
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

#ifndef MTL_CUDA_COMPRESSED2D_INCLUDE
#define MTL_CUDA_COMPRESSED2D_INCLUDE

#include <iostream>
#include <cassert>

#include <boost/numeric/mtl/cuda/config.cu>
#include <boost/numeric/mtl/cuda/compressed2D_kernel.cu>
#include <boost/numeric/mtl/cuda/get_device_value.cu>
#include <boost/numeric/mtl/cuda/vector_cuda.cu>

#define BLOCK_SIZE 512


namespace mtl { namespace cuda {
/// Class for replicating dense matrix on host and device
template <typename T>
class compressed2D
{
    typedef compressed2D<T>                self;

  public:
    typedef T                        value_type;

    /// Constructor from type T 
    compressed2D(unsigned num_rows=1, unsigned num_cols=1, const T& value= T() , bool on_host=true )
      : nnz(0),
	num_rows(num_rows),
        num_cols(num_cols),
	h_ptr(new unsigned[num_rows+1]), 
	h_indices(0), 
	h_data(0),
	d_ptr(device_vector_new<unsigned>(0, num_rows+1)),
	d_indices(device_vector_new<unsigned>(0, 0)),
	d_data(device_vector_new<T>(T(0), 0)),
        on_host(on_host)
    {
      for (unsigned i= 0; i <= num_rows; i++) h_ptr[i]= 0;
    }

    ~compressed2D() {
      // Deallocation
        delete [] h_ptr;
	delete [] h_indices;
        delete [] h_data;
	hipFree(d_ptr);
        hipFree(d_indices);
        hipFree(d_data);
    }


    T& operator()(int num_row, int num_col) {
        assert(num_row >= 0 && num_row < num_rows && num_col >= 0 && num_col < num_cols);
        to_host();
        return h_data[num_row * num_rows + num_col];
    }

    T read(int row, int col) const
    {
        assert(row >= 0 && row < num_rows && col >= 0 && col < num_cols);
        T temp;
        int num_elm;
        num_elm= h_ptr[row+1]-h_ptr[row];
//       std::cout<< "row=" << row << "  col=" << col << " num_elm=" << num_elm << "\n";
        for (unsigned i= h_ptr[row]; i < h_ptr[row]+num_elm; i++){
// 		std::cout<< "schleife i=" << i << "\n";
                if (h_indices[i] == col && on_host)
                {
                        temp= h_data[i];
// 			std::cout<< "temp=" << temp << "\n";
			break;
                }
                else if (h_indices[i] == col && !on_host)
                {
                        temp= get_device_value(&d_data[i]);
			break;
                }
                else
                {
                        temp= T(0);
                }
        }
        return temp;
    }

    T operator()(int num_row, int num_col) const { return read(num_row, num_col); }

    bool valid_host() const { return on_host; }
    bool valid_device() const { return !on_host; }
    friend int  num_cols(const self& x) { return x.num_cols; }
    friend int  num_rows(const self& x) { return x.num_rows; }
    friend int  size(const self& x) { return x.num_rows * x.num_cols; }

    template<typename Vector>
    Vector operator * (const Vector& x)
    {	
	assert(num_rows == size(x));
	Vector temp(size(x), 0);
	unsigned aux=0;
	
 	temp.on_host= !(x.valid_device() && (*this).valid_device());
	if (temp.on_host){
// 	   std::cout<< "Mat_vec_mult auf Host\n";
	   for (unsigned i= 0; i < size(x)+2*(size(x)-1); i++){
	     temp.start[aux] += h_data [ i ] * x [h_indices [ i ]];
	     if( h_ptr[aux+1]-(i+1)==0 ){
		aux++;
	     } 
	   }
	   
	} else {
 //	    std::cout<< "mat_vec_mult auf device\n";
	    temp.to_device(); // if not yet there
	    dim3 dimGrid(num_cols/BLOCK_SIZE+1), dimBlock(BLOCK_SIZE);
	    sparse_mat_vec_mult<<<dimGrid, dimBlock>>>(num_rows, d_ptr, d_indices, d_data, x.dptr, temp.dptr);
	}
	return temp;
    }

    void change_nnz(unsigned n)
    {
        nnz= n;
	if (h_indices) delete[] h_indices;
        h_indices= new unsigned [n];
	if (h_data) delete[] h_data;
        h_data= new T [n];

        d_ptr= device_vector_new<unsigned>(0, num_rows+1);
	if (d_indices) hipFree(d_indices);
        d_indices= device_vector_new<unsigned>(0, n);
	if (d_data) hipFree(d_data);
        d_data= device_vector_new<T>(T(0), n);
    }

    void laplacian_setup(unsigned m, unsigned n)
    {
	assert(m*n == num_rows); assert(num_cols == num_rows);
	change_nnz(5*m*n - 2*m - 2*n);
	
	unsigned pos= 0;
	for (unsigned i= 0; i < m; i++)
	      for (unsigned j= 0; j < n; j++) {
		  unsigned row= i * n + j;
		  h_ptr[row]= pos;
		  if (i > 0) {
		     h_indices[pos]= row-n;
		     h_data[pos++]= -1;
		  }
		  if (j > 0) {
		     h_indices[pos]= row-1;
		     h_data[pos++]= -1;
		  }
		  h_indices[pos]= row;
		  h_data[pos++]= 4;
		  if (j < n-1)  {
		     h_indices[pos]= row+1;
		     h_data[pos++]= -1;
		  } 
		  if (i < m-1) {
		     h_indices[pos]= row+n;
		     h_data[pos++]= -1;
		  }
	      }
        assert(pos == 5*m*n - 2*m - 2*n);
	h_ptr[num_rows]= pos;
	on_host= true;
    }


    void simpel_laplacian_setup(unsigned n, int d)
    {
        unsigned num=n+2*(n-1), temp=0;
        change_nnz(num);
// 	std::cout<< "n=" << n << " laplace elemente=" << num << "\n";
//         std::cout<< "simpel_laplacian_setup\n";
        h_ptr[0]= 0; 
	h_ptr[1]= 2; 
	h_indices[0]= 0; 
	h_indices[1]= 1;
        h_data[0] = T(d); 
	h_data[1] = T(-1);
	
        for (unsigned i= 2; i < num-4; i+=3) {
          h_indices[i]=  temp; 
	  h_indices[i+1]= temp+1; 
	  h_indices[i+2]= temp+2;
          h_data[i]=   T(-1); 
	  h_data[i+1]= T(d); 
	  h_data[i+2]= T(-1);
          temp++;
     //     h_ptr[temp+1]=h_ptr[temp]+3;
        }
	for (unsigned i= 2; i < num_rows+1; i++) {
// 	  std::cout << "i=" << i << "\n";
	  h_ptr[i]=h_ptr[i-1]+3;
	}
	h_ptr[num_rows]= h_ptr[num_rows-1]+2; 
	h_indices[num-2]= n-2; 
	h_indices[num-1]= n-1;
        h_data[num-1] = T(d); 
	h_data[num-2] = T(-1);

	//sending to device
        hipMemcpy(d_ptr, h_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyHostToDevice);
        hipMemcpy(d_indices, h_indices, sizeof(unsigned)*(num), hipMemcpyHostToDevice);
        hipMemcpy(d_data, h_data, sizeof(T)*(num), hipMemcpyHostToDevice);
// for (unsigned i= 0; i < num_rows+1; i++)
//         std::cout<< "i=" << i << "h_ptr[i]=" << h_ptr[i] << "\n";
// 
// for (unsigned i= 0; i < num; i++){
//         std::cout<< "i=" << i << "  h_data[i]=" << h_data[i] <<  "  i=" << i << "h_indives[i]=" << h_indices[i] << "\n";
// }
	
// 	 std::cout<< "\nData:[ ";
// 	for (unsigned i= 0; i < num; i++) {
//                std::cout<<  h_data[i] << (i==num-1 ? " " : ", ");
//         }
// 	std::cout<< "]\nCols:[ ";
// 	for (unsigned i= 0; i < num; i++) {
//                std::cout<< h_indices[i] << (i==num-1 ? " " : ", ");
//         }	
//         std::cout<< "]\nptr :[ ";
// 	for (unsigned i= 0; i < num_rows+1; i++) {
//                std::cout<< h_ptr[i] << (i==num-1 ? " " : ", ");
//         }
//         std::cout<< "]\n\n";

    }
 
    void set_to_zero()
    {
	change_nnz(1);
	std::cout<< " num_rows=" << num_rows << "\n";
        for (unsigned i= 0; i < num_rows+1; i++){       //initializing on host 
            h_ptr[i]= 0;				//sparse rows pointer
        }
	std::cout<< " nnz=" << nnz << "\n";
        for (unsigned i= 0; i < nnz; i++){	       
            h_data[i]= 0;				//initializing data
            h_indices[i]= T(0);				//initializing cols
        }
	std::cout<< " dim2=" << nnz << "\n";
        on_host= false;
   
	//wesentlich schneller                          //initializing on device
        hipMemcpy(d_ptr , &h_ptr[0], sizeof(unsigned), hipMemcpyHostToDevice);
        for (int i= 1; i < num_rows+1; i++){
            hipMemcpy(d_ptr + i, d_ptr, sizeof(unsigned), hipMemcpyDeviceToDevice);
        }
        hipMemcpy(d_indices , &h_indices[0], sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(d_data , &h_data[0], sizeof(T), hipMemcpyHostToDevice);
        for (int i= 1; i < nnz; i++){
            hipMemcpy(d_indices + i, d_indices, sizeof(unsigned), hipMemcpyDeviceToDevice);
            hipMemcpy(d_data + i, d_data, sizeof(T), hipMemcpyDeviceToDevice);
        }

    }

    void to_host() const
    {
        if (!on_host) {
            hipMemcpy(h_ptr, d_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyDeviceToHost);
            hipMemcpy(h_indices, d_indices, sizeof(unsigned)*(nnz), hipMemcpyDeviceToHost);
            hipMemcpy(h_data, d_data, sizeof(T)*(nnz), hipMemcpyDeviceToHost);
            const_cast<self*>(this)->on_host= true;
        }
    }

    void replicate_on_host() const
    {
        if (!on_host) {
           hipMemcpy(h_ptr, d_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyDeviceToHost);
           hipMemcpy(h_indices, d_indices, sizeof(unsigned)*(nnz), hipMemcpyDeviceToHost);
           hipMemcpy(h_data, d_data, sizeof(T)*(nnz), hipMemcpyDeviceToHost);
        }
    }
 
    void to_device() const
    {
        if (on_host) {
           hipMemcpy(d_ptr, h_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyHostToDevice);
           hipMemcpy(d_indices, h_indices, sizeof(unsigned)*(nnz), hipMemcpyHostToDevice);
           hipMemcpy(d_data, h_data, sizeof(T)*(nnz), hipMemcpyHostToDevice);
           const_cast<self*>(this)->on_host= false;
        }
    }

    T* get_device_pointer() { return d_ptr; }
    const T* get_device_pointer() const { return d_ptr; }

    friend std::ostream& operator<<(std::ostream& os, const self& x)
    {
        x.replicate_on_host();
        os << "{" << x.num_rows << "," << x.num_cols << (x.valid_host() ? ",host}=\n" : ",device}=\n");
        for (int i= 0; i < x.num_rows; i++){
        os << "[";
          for (int j= 0; j < x.num_cols; j++){
             os <<  x.read(i,j) << (j==x.num_cols-1 ? "]\n" : "\t");
          }
        }
         os << "\n";

	 
	 
/*	unsigned  num=x.num_cols+2*(x.num_cols-1);
	 os<< "\nData:[ ";
	for (unsigned i= 0; i < num; i++) {
               os<<  x.h_data[i] << (i==num-1 ? " " : ", ");
        }
	os<< "]\nCols:[ ";
	for (unsigned i= 0; i < num; i++) {
               os<< x.h_indices[i] << (i==num-1 ? " " : ", ");
        }	
        os<< "]\nptr :[ ";
	for (unsigned i= 0; i < x.num_rows+1; i++) {
               os<< x.h_ptr[i] << (i==num-1 ? " " : ", ");
        }
        std::cout<< "]\n\n";	 */
	 
        return os;
    }

    unsigned    nnz;
    unsigned    num_cols, num_rows;
    unsigned*   h_ptr;	   // sparse_rows on host
    unsigned*   d_ptr;	   // sparse_rows on device
    unsigned*   h_indices; // cols on host 
    unsigned*   d_indices; // cols on device
    T*     h_data;	// Value on host 
    T*     d_data;   	// Value on device (allocated as pointer whose content is referred)
    bool   on_host;

};

}} // namespace mtl::cuda

#endif // MTL_CUDA_COMPRESSED2D_INCLUDE
