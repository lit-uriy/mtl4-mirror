#include "hip/hip_runtime.h"
// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University. 
//               2008 Dresden University of Technology and the Trustees of Indiana University. 
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

#ifndef MTL_CUDA_DOT_KERNEL_INCLUDE
#define MTL_CUDA_DOT_KERNEL_INCLUDE

#include <boost/numeric/mtl/cuda/cuda_dot_kernel.h>

namespace mtl { namespace cuda {

template <typename T, unsigned blocksize>
__global__ void reduce_kernel_kompliziert(T* out, T* in, unsigned int n)
{
    extern __shared__ T sdata[];

    //all threads load one element to shared memory
    unsigned int id= threadIdx.x,
	         i = blockIdx.x * blocksize * 2 + id,
	         gridSize = blocksize * 2 * gridDim.x;
    sdata[id]= 0;
    while (i < n){
	sdata[id]+= in[i] + in[i+blocksize];
	i += gridSize;
    }
    __syncthreads();
    
    //reduction in shared memory
    if (blocksize >= 512) {
	if (id < 256) sdata[id]+= sdata[id + 256];
	__syncthreads();
    }
    if (blocksize >= 256) {
	if (id < 128) sdata[id]+= sdata[id + 128];
	__syncthreads();
    }
    if (blocksize >= 128) {
	if (id < 64) sdata[id]+= sdata[id + 64];
	__syncthreads();
    }
    if (id < 32){
	if (blocksize >= 64) sdata[id]+= sdata[id + 32];
	if (blocksize >= 32) sdata[id]+= sdata[id + 16];
	if (blocksize >= 16) sdata[id]+= sdata[id +  8];
	if (blocksize >=  8) sdata[id]+= sdata[id +  4];
	if (blocksize >=  4) sdata[id]+= sdata[id +  2];
	if (blocksize >=  2) sdata[id]+= sdata[id +  1];
    }
    //write result of block to global memory
    if (id == 0) out[blockIdx.x]= sdata[0];

}

template <typename T>
__global__ void reduce_kernel(T* out, T* in, unsigned int n)
{
    extern __shared__ T sdata[];
    unsigned int id= threadIdx.x,i= blockDim.x * gridDim.x + id;

    if (i < n)
	sdata[i]= in[i];
    else
	sdata[i]= 0;

    int j= 1;
    while (j < n) j<<= 1;
    j>>= 1;

    for (; j > 32; j>>= 1) {
	sdata[i]+= sdata[i + j];
	__syncthreads();
    }

    for (; j > 0; j>>= 1)
	sdata[i]+= sdata[i + j];

    //write result of block to global memory
    if (id == 0) out[blockIdx.x]= sdata[0];

}

template <typename T>
__global__ void dot_kernel(T* out, T* v1, T* v2, unsigned n)
{
    extern __shared__ T sdata[];

    //all threads load one element to shared memory
    unsigned int id= threadIdx.x,
	         i = blockIdx.x * gridDim.x + id,
	         step= blockIdx.x * gridDim.x;
    
    sdata[id]= 0;

    //do multiplication parallel on gpu not on cpu
    // divide into blocks of n/step

    unsigned blocks= n / step, nn= blocks * step, rem= n - nn;
    for (int j= id; j < nn; j+= steps)
	sdata[id]+= v1[j] * v2[j];

    if (nn + id < n)
	sdata[id]+= v1[nn + id] * v2[nn + id];

}



}} // namespace mtl::cuda

#endif // MTL_CUDA_DOT_KERNEL_INCLUDE
