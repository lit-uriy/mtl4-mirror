#include "hip/hip_runtime.h"
// Software License for MTL
//
// Copyright (c) 2007 The Trustees of Indiana University.
//               2008 Dresden University of Technology and the Trustees of Indiana University.
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
//
// This file is part of the Matrix Template Library
//
// See also license.mtl.txt in the distribution.

#include <iostream>
#include <complex>
#include <cmath>

#include <boost/numeric/mtl/cuda/vector_cuda.cu>
#include <boost/numeric/mtl/cuda/dot.cu>


template <typename VectorU>
void test(VectorU& u, VectorU& v, const char* name)
{
    //using mtl::vector::dot;
    typedef typename mtl::Collection<VectorU>::size_type  size_type;
    for (size_type i= 0; i < size(v); i++)
	u[i]= i+1, v[i]= i+1;

    std::cout << name << "\n dot(u, v) = " << dot(u, v) << "\n"; std::cout.flush();
    if (std::abs(dot(u, v) - 285.0) > 0.01) throw "dot product wrong";
}
 



int main( int argc, char** argv)
{
    const int size= 9;

    mtl::cuda::vector<int>     i(size), j(size);
    mtl::cuda::vector<float>   u(size), v(size), w(size);
    mtl::cuda::vector<double>  x(size), y(size), z(size);
  //  mtl::cuda::vector<std::complex<double> >  xc(size), yc(size), zc(size);

    test(i, j, "test int");
    test(u, v, "test float");
    test(x, y, "test double");
    // test(xc, yc, "test complex<double>");


//     mtl::cuda::vector_cuda<float, parameters<mtl::row_major> >   ur(size), vr(size), wr(size);
//     test(ur, vr, "test float in row vector");

    return 0;
}
