// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University.
//               2008 Dresden University of Technology and the Trustees of Indiana University.
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

#include <iostream>
#include <boost/numeric/mtl/cuda/cg.cu>
#include <boost/numeric/mtl/cuda/config.cu>
#include <boost/numeric/mtl/cuda/dense2D.cu>
#include <boost/numeric/mtl/cuda/dot.cu>
#include <boost/numeric/mtl/cuda/compressed2D.cu>
#include <boost/numeric/mtl/cuda/scalar.cu>
#include <boost/numeric/mtl/cuda/vector_cuda.cu>


int main(int argc, char* argv[])
{
  using namespace mtl;

  // For a more realistic example set size to 1000 or larger
  const int size = 1000, N = size * size;
  int iter=0;
  mtl::cuda::activate_best_gpu(); 
  typedef mtl::cuda::compressed2D<double>  matrix_type;
  
  matrix_type         A(N, N);
  A.laplacian_setup(size, size); A.to_device();

  mtl::cuda::vector<double> x(N, 1), b(N), r(N);
  x.to_device();
   
  b = A * x;
  x= 0;

  double toleranz=   0.0000001;
  int	 iterations= 1000;
  
  iter=cg(A, x, b, iterations, toleranz);
//      std::cout<< "x=" << x << "\n";
  //r=b- A*x;
//       std::cout<< "r=" << r << "\n";
    std::cout <<(iter==iterations ? "Total  Iterations: " : "problem solved in the iteration: ")<<iter<<"\n";
  
  

  return 0;
}
