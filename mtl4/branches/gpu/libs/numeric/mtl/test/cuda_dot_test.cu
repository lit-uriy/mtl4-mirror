#include "hip/hip_runtime.h"
// Software License for MTL
//
// Copyright (c) 2007 The Trustees of Indiana University.
//               2008 Dresden University of Technology and the Trustees of Indiana University.
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
//
// This file is part of the Matrix Template Library
//
// See also license.mtl.txt in the distribution.

#include <iostream>
#include <complex>
#include <cmath>

#include <boost/numeric/mtl/cuda/vector_cuda.cu>
#include <boost/numeric/mtl/cuda/dot.cu>


template <typename VectorU>
void test(VectorU& u, VectorU& v, const char* name)
{
    //using mtl::vector::dot;
    typedef typename mtl::Collection<VectorU>::size_type  size_type;
    typedef typename mtl::Collection<VectorU>::value_type value_type;
    
    for (size_type i= 0; i < size(v); i++)
	u[i]= i, v[i]= size(v)-i;
    value_type temp(0), temp2(0);
    u.to_host(); v.to_host(); 
    for (size_type i= 0; i < size(v); i++)
	temp+= u[i] * v[i];
    for (int i= size(v)-1; i >= 0; i--)
 	temp2+= u[i] * v[i];
   
    std::cout << name << " size = " << size(v) << "\n dot(u, v) = " << dot(u, v) << "\n"; std::cout.flush();
    std::cout << "temp == " << temp << ", temp2 == " << temp2 << "\n";
    std::cout << std::abs(dot(u, v) - temp) << "\n";
    if (std::abs(dot(u, v) - temp) > 0.1 * abs(temp)) throw "dot product wrong";
}
 



int main( int argc, char** argv)
{
    const int size= 1029;

    mtl::cuda::vector<int>     i(size), j(size);
    mtl::cuda::vector<float>   u(size), v(size), w(size);
    mtl::cuda::vector<double>  x(size), y(size), z(size);
  //  mtl::cuda::vector<std::complex<double> >  xc(size), yc(size), zc(size);

  /*
    test(i, j, "test int");
    test(u, v, "test float"); */
    for (int k= 10; k < 30000000; k*= 3) {
        mtl::cuda::vector<double>     v(k), w(k);
	test(v, w, "test sizes");
    }
//    test(x, y, "test double");
    // test(xc, yc, "test complex<double>");


//     mtl::cuda::vector_cuda<float, parameters<mtl::row_major> >   ur(size), vr(size), wr(size);
//     test(ur, vr, "test float in row vector");

    return 0;
}
