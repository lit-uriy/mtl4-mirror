// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University. 
//               2008 Dresden University of Technology and the Trustees of Indiana University. 
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

#ifndef MTL_CUDA_COMPRESSED2D_INCLUDE
#define MTL_CUDA_COMPRESSED2D_INCLUDE

#include <iostream>
#include <cassert>

#include <boost/numeric/mtl/cuda/config.cu>
#include <boost/numeric/mtl/cuda/compressed2D_kernel.cu>
#include <boost/numeric/mtl/cuda/get_device_value.cu>
#include <boost/numeric/mtl/cuda/vector_cuda.cu>

#define BLOCK_SIZE 512


namespace mtl { namespace cuda {
/// Class for replicating dense matrix on host and device
template <typename T>
class compressed2D
{
    typedef compressed2D<T>                self;

  public:
    typedef T                        value_type;

    /// Constructor from type T 
    compressed2D(unsigned num_rows=1, unsigned num_cols=1, const T& value= T() , bool on_host=true )
      : num_rows(num_rows),
        num_cols(num_cols),
        on_host(on_host)
    {
      
    }

    ~compressed2D() {
      // Deallocation
        delete [] h_ptr;
	delete [] h_indices;
        delete [] h_data;
	hipFree(d_ptr);
        hipFree(d_indices);
        hipFree(d_data);
    }


    T& operator()(int num_row, int num_col) {
        assert(num_row >= 0 && num_row < num_rows && num_col >= 0 && num_col < num_cols);
        to_host();
        return h_data[num_row * num_rows + num_col];
    }

    T read(int row, int col) const
    {
        assert(row >= 0 && row < num_rows && col >= 0 && col < num_cols);
        T temp;
        int num_elm;
        num_elm= h_ptr[row+1]-h_ptr[row];
//       std::cout<< "row=" << row << "  col=" << col << " num_elm=" << num_elm << "\n";
        for (unsigned i= h_ptr[row]; i < h_ptr[row]+num_elm; i++){
// 		std::cout<< "schleife i=" << i << "\n";
                if (h_indices[i] == col && on_host)
                {
                        temp= h_data[i];
// 			std::cout<< "temp=" << temp << "\n";
			break;
                }
                else if (h_indices[i] == col && !on_host)
                {
                        temp= get_device_value(&d_data[i]);
			break;
                }
                else
                {
                        temp= T(0);
                }
        }
        return temp;
    }

    T operator()(int num_row, int num_col) const { return read(num_row, num_col); }

    bool valid_host() const { return on_host; }
    bool valid_device() const { return !on_host; }
    friend int  num_cols(const self& x) { return x.num_cols; }
    friend int  num_rows(const self& x) { return x.num_rows; }
    friend int  size(const self& x) { return x.num_rows * x.num_cols; }

    template<typename Vector>
    Vector operator * (const Vector& x)
    {	
	assert(num_rows == size(x));
	Vector temp(size(x), 0);
	unsigned aux=0;
	
 	temp.on_host= !(x.valid_device() && (*this).valid_device());
	if (temp.on_host){
	   std::cout<< "Mat_vec_mult auf Host\n";
   
	   for (unsigned i= 0; i < size(x)+2*(size(x)-1); i++){
	     temp.start[aux] += h_data [ i ] * x [h_indices [ i ]];

	     if( h_ptr[aux+1]-(i+1)==0 ){
	     aux++;
	     } 
   
	   }
	   
	} else {
 	    std::cout<< "mat_vec_mult auf device\n";
	    temp.to_device(); // if not yet there
	    dim3 dimGrid(num_cols/BLOCK_SIZE+1), dimBlock(BLOCK_SIZE);
 	    std::cout<< "num_cols/BLOCK_SIZE=" << num_cols/BLOCK_SIZE+1 << "\n";
	    sparse_mat_vec_mult<<<dimGrid, dimBlock>>>(num_rows, d_ptr, d_indices, d_data, x.dptr, temp.dptr);
	}
	return temp;
    }

    void change_dim(unsigned n)
    {
        dim= n;
        h_ptr= new unsigned [num_rows+1];
        h_indices= new unsigned [n];
        h_data= new T [n];

        d_ptr= device_vector_new<unsigned>(0, num_rows+1);
        d_indices= device_vector_new<unsigned>(0, n);
        d_data= device_vector_new<T>(T(0), n);
        //set_to_zero();  // not necessary
    }

    void simpel_laplacian_setup(unsigned n, int d)
    {
        unsigned num=n+2*(n-1), temp=0;
        change_dim(num);
	
        std::cout<< "simpel_laplacian_setup\n";
        h_ptr[0]= 0; 
	h_ptr[1]= 2; 
	h_indices[0]= 0; 
	h_indices[1]= 1;
        h_data[0] = T(d); 
	h_data[1] = T(-1);
	
        for (unsigned i= 2; i < num-4; i+=3) {
          h_indices[i]=  temp; 
	  h_indices[i+1]= temp+1; 
	  h_indices[i+2]= temp+2;
          h_data[i]=   T(-1); 
	  h_data[i+1]= T(d); 
	  h_data[i+2]= T(-1);
          temp++;
          h_ptr[temp+1]=h_ptr[temp]+3;
        }
        
	h_ptr[num_rows]= h_ptr[num_rows-1]+2; 
	h_indices[num-2]= n-2; 
	h_indices[num-1]= n-1;
        h_data[num-1] = T(d); 
	h_data[num-2] = T(-1);

	//sending to device
        hipMemcpy(d_ptr, h_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyHostToDevice);
        hipMemcpy(d_indices, h_indices, sizeof(unsigned)*(num), hipMemcpyHostToDevice);
        hipMemcpy(d_data, h_data, sizeof(T)*(num), hipMemcpyHostToDevice);

        on_host= false;

	
// 	 std::cout<< "\nData:[ ";
// 	for (unsigned i= 0; i < num; i++) {
//                std::cout<<  h_data[i] << (i==num-1 ? " " : ", ");
//         }
// 	std::cout<< "]\nCols:[ ";
// 	for (unsigned i= 0; i < num; i++) {
//                std::cout<< h_indices[i] << (i==num-1 ? " " : ", ");
//         }	
//         std::cout<< "]\nptr :[ ";
// 	for (unsigned i= 0; i < num_rows+1; i++) {
//                std::cout<< h_ptr[i] << (i==num-1 ? " " : ", ");
//         }
//         std::cout<< "]\n\n";

    }
 
    void set_to_zero()
    {
	change_dim(1);
	std::cout<< " num_rows=" << num_rows << "\n";
        for (unsigned i= 0; i < num_rows+1; i++){       //initializing on host 
            h_ptr[i]= 0;				//sparse rows pointer
        }
	std::cout<< " dim=" << dim << "\n";
        for (unsigned i= 0; i < dim; i++){	       
            h_data[i]= 0;				//initializing data
            h_indices[i]= T(0);				//initializing cols
        }
	std::cout<< " dim2=" << dim << "\n";
        on_host= false;
   
	//wesentlich schneller                          //initializing on device
        hipMemcpy(d_ptr , &h_ptr[0], sizeof(unsigned), hipMemcpyHostToDevice);
        for (int i= 1; i < num_rows+1; i++){
            hipMemcpy(d_ptr + i, d_ptr, sizeof(unsigned), hipMemcpyDeviceToDevice);
        }
        hipMemcpy(d_indices , &h_indices[0], sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(d_data , &h_data[0], sizeof(T), hipMemcpyHostToDevice);
        for (int i= 1; i < dim; i++){
            hipMemcpy(d_indices + i, d_indices, sizeof(unsigned), hipMemcpyDeviceToDevice);
            hipMemcpy(d_data + i, d_data, sizeof(T), hipMemcpyDeviceToDevice);
        }

    }

    void to_host() const
    {
        if (!on_host) {
            hipMemcpy(h_ptr, d_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyDeviceToHost);
            hipMemcpy(h_indices, d_indices, sizeof(unsigned)*(dim), hipMemcpyDeviceToHost);
            hipMemcpy(h_data, d_data, sizeof(T)*(dim), hipMemcpyDeviceToHost);
            const_cast<self*>(this)->on_host= true;
        }
    }

    void replicate_on_host() const
    {
        if (!on_host) {
           hipMemcpy(h_ptr, d_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyDeviceToHost);
           hipMemcpy(h_indices, d_indices, sizeof(unsigned)*(dim), hipMemcpyDeviceToHost);
           hipMemcpy(h_data, d_data, sizeof(T)*(dim), hipMemcpyDeviceToHost);
        }
    }
 
    void to_device() const
    {
        if (on_host) {
           hipMemcpy(d_ptr, h_ptr, sizeof(unsigned)*(num_rows+1), hipMemcpyHostToDevice);
           hipMemcpy(d_indices, h_indices, sizeof(unsigned)*(dim), hipMemcpyHostToDevice);
           hipMemcpy(d_data, h_data, sizeof(T)*(dim), hipMemcpyHostToDevice);
           const_cast<self*>(this)->on_host= false;
        }
    }

    T* get_device_pointer() { return d_ptr; }
    const T* get_device_pointer() const { return d_ptr; }

    friend std::ostream& operator<<(std::ostream& os, const self& x)
    {
        x.replicate_on_host();
        os << "{" << x.num_rows << "," << x.num_cols << (x.valid_host() ? ",host}=\n" : ",device}=\n");
        for (int i= 0; i < x.num_rows; i++){
        os << "[";
          for (int j= 0; j < x.num_cols; j++){
             os <<  x.read(i,j) << (j==x.num_cols-1 ? "]\n" : "\t");
          }
        }
         os << "\n";

	 
	 
/*	unsigned  num=x.num_cols+2*(x.num_cols-1);
	 os<< "\nData:[ ";
	for (unsigned i= 0; i < num; i++) {
               os<<  x.h_data[i] << (i==num-1 ? " " : ", ");
        }
	os<< "]\nCols:[ ";
	for (unsigned i= 0; i < num; i++) {
               os<< x.h_indices[i] << (i==num-1 ? " " : ", ");
        }	
        os<< "]\nptr :[ ";
	for (unsigned i= 0; i < x.num_rows+1; i++) {
               os<< x.h_ptr[i] << (i==num-1 ? " " : ", ");
        }
        std::cout<< "]\n\n";	 */
	 
        return os;
    }

    unsigned    dim;
    unsigned    num_cols, num_rows;
    unsigned*   h_ptr;	   // sparse_rows on host
    unsigned*   d_ptr;	   // sparse_rows on device
    unsigned*   h_indices; // cols on host 
    unsigned*   d_indices; // cols on device
    T*     h_data;	// Value on host 
    T*     d_data;   	// Value on device (allocated as pointer whose content is referred)
    bool   on_host;

};

}} // namespace mtl::cuda

#endif // MTL_CUDA_COMPRESSED2D_INCLUDE
