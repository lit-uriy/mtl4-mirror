// Software License for MTL
// 
// Copyright (c) 2007 The Trustees of Indiana University. 
//               2008 Dresden University of Technology and the Trustees of Indiana University. 
// All rights reserved.
// Authors: Peter Gottschling and Andrew Lumsdaine
// 
// This file is part of the Matrix Template Library
// 
// See also license.mtl.txt in the distribution.

#ifndef MTL_CUDA_VECTOR_INCLUDE
#define MTL_CUDA_VECTOR_INCLUDE
//for testing only



#include <iostream>
#include <cassert>

#include <boost/numeric/mtl/cuda/config.hpp>
#include <boost/numeric/mtl/cuda/get_device_value.cu>
#include <boost/numeric/mtl/cuda/device_vector_new.cu>
#include <boost/numeric/mtl/cuda/vector_kernel.cu>

namespace mtl { namespace cuda {

/// Class for replicating vectors on host and device
template <typename T>
class vector
{
    typedef vector<T>                self;
  public:
    typedef T                        value_type;

    /// Constructor from type T 
    vector(int n=1, const T& value= T(), bool on_host=true ) 
      : dim(n), start(new T[n]), dptr(device_vector_new<T>(n)), on_host(on_host) 
    { *this= value; } 

    ~vector() {
	 delete [] start; 
	 hipFree(dptr);
    }


    //template<typename T>
    vector<T>(const vector<T> &that){   //that Konstruktor
	dim= that.dim;
	start= new T[dim];
	on_host= that.on_host;
	if (on_host)
	    for (int i= 0; i < dim; i++)
		start[i]= that.start[i];
	else
	    hipMemcpy(dptr, that.dptr, dim*sizeof(T), hipMemcpyDeviceToDevice);
    }

    //template<typename T>
    self& operator=(const self& that)
    {
	std::cout<< "x= y zuweisung\n";
	std::cout<< "that.on_host="<< that.on_host << "\n";
	assert(dim == that.dim);

	if (this != &that) {  //unnoetige Zuweisung vermeiden
	    on_host= that.on_host;
	    if (on_host) {
		for (int i= 0; i < dim; i++)
		    start[i]= that.start[i];
	    } else
		hipMemcpy(dptr, that.dptr, dim*sizeof(T), hipMemcpyDeviceToDevice);
	}
	return *this;
    }

    // Expensive !!!
    template <typename U>
    self& operator=(const vector<U>& that)
    {
	that.replicate_on_host();
	on_host= true;
	for (int i= 0; i < dim; i++)
	    start[i]= that.start[i];
	return *this;
    }


    template <typename U>
    self& operator=(const U& src)
    {	
	std::cout<< "x=wert zuweisung\n";
        for (int i= 0; i < dim; i++) 
            start[i]= src;
	if (!on_host) { on_host= true; to_device(); }
	return *this;
    }

    template <typename U>
    self& operator*=(const U& src)
    {
        std::cout<< "x*= wert zuweisung\n";
	if (on_host && dim < host_limit) {
	    std::cout<< "on host\n";
	    for (int i= 0; i < dim; i++) 
		start[i]*= src;
	} else {
	    std::cout<< "on device\n";
	    to_device(); // if not yet there
	    dim3 dimGrid(1), dimBlock(dim); 
	    vec_rscale_asgn<value_type> sc(src, dptr);
	    launch_function<<<dimGrid, dimBlock>>>(sc);
	}
        return *this;
    }

    T& operator[](int index) {
	assert(index >= 0 && index < dim);
	to_host();
	return start[index];
    }

    T operator[](int i) const 
    {
        assert(i >= 0 && i < dim);
	return on_host ? start[i] : get_device_value(dptr + i);
    }

    bool valid_host() const { return on_host; }
    bool valid_device() const { return !on_host; }
    int  size() const { return dim; }

    void to_host()
    {
	if (!on_host) {
	    hipMemcpy(start, dptr, sizeof(T)*dim, hipMemcpyDeviceToHost);
	    on_host= true;
	}
    }

    void replicate_on_host() const
    {
	if (!on_host) 
	    hipMemcpy(const_cast<self*>(this)->start, dptr, sizeof(T)*dim, hipMemcpyDeviceToHost);
    }

    void to_device()
    {
	if (on_host) {
	    hipMemcpy(dptr, start, sizeof(T)*dim, hipMemcpyHostToDevice);
	    on_host= false;
	}
	for (int i= 0; i < dim; i++) 
            start[i]= 77;
    }
    
    friend std::ostream& operator<<(std::ostream& os, self& x)
    {
	x.replicate_on_host();
	os << "{" << x.size() << (x.valid_host() ? ",host}(" : ",device}(");
	for (int i= 0; i < x.size(); i++)
	    os << x.start[i] << (i < x.dim - 1 ? ", " : ")");
	return os;
    }

  
    int  dim;
    T*   start; // Value on host //TODO    malloc sizeof(T)*dim
    T*   dptr;   // Value on device (allocated as pointer whose content is referred)
    bool on_host;
};

}} // namespace mtl::cuda

#endif // MTL_CUDA_VECTOR_INCLUDE
